
#include <hip/hip_runtime.h>
/*
    Perform a "half tomography" sinogram conversion.
    A 360 degrees sinogram is converted to a 180 degrees sinogram with a
    field of view extended (at most) twice".
    *
    Parameters:
    * sinogram: the 360 degrees sinogram, shape (n_angles, n_x)
    * output: the 160 degrees sinogram, shape (n_angles/2, rotation_axis_position * 2)
    * weights: an array of weight, size n_x - rotation_axis_position
*/
__global__ void halftomo_kernel(
    float* sinogram,
    float* output,
    float* weights,
    int n_angles,
    int n_x,
    int rotation_axis_position
) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int n_a2 = (n_angles + 1) / 2;
    int d = n_x - rotation_axis_position;
    int n_x2  = 2 * rotation_axis_position;
    int r = rotation_axis_position;

    if ((x >= n_x2) || (y >= n_a2)) return;

    // output[:, :r - d] = sino[:n_a2, :r - d]
    if (x < r - d) {
        output[y * n_x2 + x] = sinogram[y * n_x + x];
    }

    // output[:, r-d:r+d] = (1 - weights) * sino[:n_a2, r-d:]
    else if (x < r+d) {
        float w = weights[x - (r - d)];
        output[y * n_x2 + x] = (1.0f - w) * sinogram[y*n_x + x] \
                                   + w * sinogram[(n_a2 + y)*n_x + (n_x2 - 1 - x)];
    }

    // output[:, nx:] = sino[n_a2:, ::-1][:, 2 * d :] = sino[n_a2:, -2*d-1:-n_x-1:-1]
    else {
        output[y * n_x2 + x] = sinogram[(n_a2 + y)*n_x + (n_x2 - 1 - x)];
    }

}

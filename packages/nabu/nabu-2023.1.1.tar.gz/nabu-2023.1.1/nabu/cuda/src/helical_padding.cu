
#include <hip/hip_runtime.h>

// see  nabu/pipeline/helical/filtering.py for details

__global__ void padding(
    float* data,
    int* mirror_indexes,
    
#if defined(MIRROR_CONSTANT_VARIABLE_ROT_POS) || defined(MIRROR_EDGES_VARIABLE_ROT_POS)
    int *rot_axis_pos,
#else
    int rot_axis_pos,    
#endif
    int Nx,
    int Ny,
    int Nx_padded,
    int pad_left_len,
    int pad_right_len
#if defined(MIRROR_CONSTANT) || defined(MIRROR_CONSTANT_VARIABLE_ROT_POS)
    ,float pad_left_val,
    float pad_right_val    
#endif
) {
  
  int x = blockDim.x * blockIdx.x + threadIdx.x;
  int y = blockDim.y * blockIdx.y + threadIdx.y;
    
  if ((x >= Nx_padded) || (y >= Ny) || x < Nx) return;
    
  int idx = y*Nx_padded  +  x;

  int y_mirror = mirror_indexes[y];
  
  int x_mirror =0 ; 
 
#if defined(MIRROR_CONSTANT_VARIABLE_ROT_POS) || defined(MIRROR_EDGES_VARIABLE_ROT_POS)
  int two_rots = rot_axis_pos[y] + rot_axis_pos[y_mirror];
#else
  int two_rots = 2*rot_axis_pos ;
#endif

  if( two_rots > Nx)  {
    x_mirror = two_rots - x ;
    if (x_mirror  < 0 ) {
#if defined(MIRROR_CONSTANT) || defined(MIRROR_CONSTANT_VARIABLE_ROT_POS)
      if( x < Nx_padded - pad_left_len) {
	data[idx] = pad_left_val;
      } else {
	data[idx] = pad_right_val; 
      }
#else
      if( x < Nx_padded - pad_left_len) {
	data[idx] = data[y_mirror*Nx_padded  + 0];
      } else {
	data[idx] = data[y*Nx_padded  +  0];
      }
#endif

    } else {
      data[idx] = data[y_mirror*Nx_padded  +  x_mirror];
    }
  } else {
    x_mirror = two_rots - (x - Nx_padded) ;
    if (x_mirror  > Nx-1 ) {
#if defined(MIRROR_CONSTANT) || defined(MIRROR_CONSTANT_VARIABLE_ROT_POS)
      if( x < Nx_padded - pad_left_len) {
	data[idx] =  pad_left_val ;
      } else {
	data[idx] = pad_right_val;
      }
#else
      if( x < Nx_padded - pad_left_len) {
	data[idx] = data[y*Nx_padded  + Nx - 1 ];
      } else {
	data[idx] = data[y_mirror*Nx_padded  +  Nx-1];
      }
#endif

    } else {
      data[idx] = data[y_mirror*Nx_padded  +  x_mirror];
    }
  }
  return;
}

